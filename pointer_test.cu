#include "hip/hip_runtime.h"
#include <stdio.h>

#include "utility/vec.hpp"

using foo_t = int (*) ();

__device__ int ret_1() {
	return 1;
}

__device__ foo_t p_ret_1 = ret_1;

__global__ void call(foo_t foo) {
	printf("%d\n", foo());
}

int main() {
	printf("Hello Ray Marching!\n");
	hipError_t err;
	
	foo_t ret_1_k = ret_1;
    err = hipMemcpyFromSymbol(&ret_1_k, HIP_SYMBOL(p_ret_1), sizeof(foo_t));
	printf("%s\n", hipGetErrorName(err));
	
	call<<<1, 1>>>(ret_1_k);
	
	hipDeviceSynchronize();
	
	err = hipGetLastError();
	printf("%s\n", hipGetErrorName(err));
	
	return 0;
}